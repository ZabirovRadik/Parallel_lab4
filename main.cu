#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <filesystem>
#include <string>
#include <cstdlib>
#include <ctime>
#include <iomanip>
#include <hip/hip_runtime.h>

namespace fs = std::filesystem;

using namespace std;

size_t XorShift128() {
    static size_t x = 123456789, y = 362436069, z = 521288629, w = 88675123;
    size_t t = x ^ (x << 11);
    x = y; y = z; z = w;
    w = (w ^ (w >> 19)) ^ (t ^ (t >> 8));
    return w % 1000;
}

void save_matrix(const vector<vector<int>>& mat, const fs::path& dir, const string& filename) {
    if (!fs::exists(dir)) {
        fs::create_directories(dir);
    }
    fs::path filepath = dir / (filename + ".txt");
    ofstream file(filepath);
    if (!file) {
        throw runtime_error("Can't open file for writing: " + filepath.string());
    }
    for (const auto& row : mat) {
        for (int v : row) {
            file << v << ' ';
        }
        file << "\n";
    }
}

void save_result(const vector<vector<int>>& result, int num_threads, int size) {
    fs::path dir = fs::path("data") / ("threads_" + to_string(num_threads));
    save_matrix(result, dir, "multiplied_" + to_string(size));
}

void save_report(const string& report) {
    fs::path dir = "data";
    if (!fs::exists(dir)) {
        fs::create_directories(dir);
    }
    ofstream file(dir / "results.txt", ios::app);
    if (!file) {
        throw runtime_error("Can't open report file for writing");
    }
    file << report << "\n";
}

vector<vector<int>> load_matrix(int size, const string& filename) {
    fs::path filepath = fs::path("data") / to_string(size) / (filename + ".txt");
    ifstream file(filepath);
    if (!file) throw runtime_error("Cannot open file: " + filepath.string());

    vector<vector<int>> mat(size, vector<int>(size));
    for (int i = 0; i < size; ++i)
        for (int j = 0; j < size; ++j)
            file >> mat[i][j];

    return mat;
}

__global__ void matrixMultiplyKernel(int* a, int* b, int* c, int size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < size && col < size) {
        int sum = 0;
        for (int k = 0; k < size; k++) {
            sum += a[row * size + k] * b[k * size + col];
        }
        c[row * size + col] = sum;
    }
}

vector<vector<int>> multiplyMatricesCUDA(const vector<vector<int>>& a, 
                                      const vector<vector<int>>& b, 
                                      int threadsPerBlock) {
    int size = a.size();
    vector<vector<int>> result(size, vector<int>(size, 0));

    int* h_a = new int[size * size];
    int* h_b = new int[size * size];
    int* h_c = new int[size * size];

    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            h_a[i * size + j] = a[i][j];
            h_b[i * size + j] = b[i][j];
        }
    }

    int* d_a, * d_b, * d_c;
    hipMalloc(&d_a, size * size * sizeof(int));
    hipMalloc(&d_b, size * size * sizeof(int));
    hipMalloc(&d_c, size * size * sizeof(int));

    hipMemcpy(d_a, h_a, size * size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size * size * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlockDim(threadsPerBlock, threadsPerBlock);
    dim3 blocksPerGrid((size + threadsPerBlock - 1) / threadsPerBlock, 
                      (size + threadsPerBlock - 1) / threadsPerBlock);

    matrixMultiplyKernel<<<blocksPerGrid, threadsPerBlockDim>>>(d_a, d_b, d_c, size);

    hipMemcpy(h_c, d_c, size * size * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            result[i][j] = h_c[i * size + j];
        }
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    return result;
}

int main() {
    srand(time(0));
    bool use_existing = false;

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        cerr << "ERROR: No CUDA devices found" << endl;
        return 1;
    }
    cout << "Found CUDA devices: " << deviceCount << endl;

    vector<int> sizes = {100, 200, 300, 400, 500, 600};
    vector<int> thread_counts = {2, 4, 8, 12, 16};

    for (int size : sizes) {
        vector<vector<int>> A(size, vector<int>(size));
        vector<vector<int>> B(size, vector<int>(size));
        
        if (use_existing) {
            A = load_matrix(size, "A");
            B = load_matrix(size, "B");
        }
        else {
            for (int i = 0; i < size; ++i) {
                for (int j = 0; j < size; ++j) {
                    A[i][j] = XorShift128();
                    B[i][j] = XorShift128();
                }
            }
            save_matrix(A, fs::path("data") / to_string(size), "A");
            save_matrix(B, fs::path("data") / to_string(size), "B");
        }

        cout << "\nProcessing matrices " << size << "x" << size << "..." << endl;

        for (int threads : thread_counts) {
            cout << "Threads: " << threads << endl;
            
            auto start = chrono::high_resolution_clock::now();
            auto result = multiplyMatricesCUDA(A, B, threads);
            auto end = chrono::high_resolution_clock::now();

            save_result(result, threads, size);

            auto duration = chrono::duration_cast<chrono::milliseconds>(end - start);
            string report = "Size: " + to_string(size) + 
                          ", Threads: " + to_string(threads) + 
                          ", Time: " + to_string(duration.count()) + " ms";
            save_report(report);
            
            cout << report << endl;
        }
    }

    return 0;
}